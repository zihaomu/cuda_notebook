#include <iostream>
#include <hip/hip_runtime.h>

#include "cuda_gemm_utils.cuh"

void check_cuda(hipError_t err, const char* const func, const char* const file,
    const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
            << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void check_cuda_last(const char* const file, const int line)
{
    hipError_t const err{hipGetLastError()};
    
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
            << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
